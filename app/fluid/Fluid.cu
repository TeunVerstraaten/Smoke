#include "hip/hip_runtime.h"
#include "../math/Math.h"
#include "Fluid.cuh"

#include <cassert>
#include <cstdint>
#include <vector>

namespace app::fluid {

    // See Jos Stam's paper "Real-Time Fluid Dynamics for Games" for the algorithm below

    static const unsigned g_threads = 1024;
    static const unsigned g_blocks  = (g_point_count * g_point_count + g_threads - 1) / g_threads;
    __constant__ unsigned g_n       = g_point_count * g_point_count;

    struct Index {
        uint16_t i;
        uint16_t j;
    };

    /******************** Static function declarations ************************/
    static Index screen_to_array_indices(float x, float y);
    static float clamp_to_zero_one(float val);
    static void  advect(float* current, const float* previous, const float* u, const float* v);
    static void  diffuse(float* current, const float* previous, float weight);
    static void  add_scaled(float* target, const float* addend, float multiplier);
    static void  project(float* u_gpu, float* v_gpu, float* u_previous_gpu, float* v_previous_gpu);

    /******************** CUDA kernel function declarations ************************/
    __global__ void add_around_point_kernel(float* density, uint16_t i_target, uint16_t j_target, float multiplier, float radius);
    __global__ void set_to_zero_kernel(float* data);
    __global__ void decrease_density_kernel(float* density);
    __global__ void advect_kernel(float* current, const float* previous, const float* u, const float* v, float ratio);
    __global__ void set_bounds_to_zero_kernel(float* data);
    __global__ void diffuse_kernel(float* current, const float* previous, float ratio);
    __global__ void add_scaled_kernel(float* target, const float* addend, float multiplier);
    __global__ void project_step_3_kernel(float* u_current, float* v_current, const float* u_previous);
    __global__ void project_step_2_kernel(float* u_previous);
    __global__ void project_step_1_kernel(float* v_previous, const float* u_current, const float* v_current);
    __device__ float    distance_kernel(uint16_t i_1, uint16_t j_1, uint16_t i_2, uint16_t j_2);
    __device__ float    sum_neighbors_kernel(const float* data, uint16_t i, uint16_t j);
    __device__ float    horizontal_difference(const float* data, uint16_t i, uint16_t j);
    __device__ float    vertical_difference(const float* source, uint16_t i, uint16_t j);
    __device__ uint32_t to_linear_index(uint16_t i, uint16_t j);

    /******************** Fluid member function implementations ************************/
    Fluid::Fluid() {
        hipMalloc(&m_density_cuda, g_point_count * g_point_count * sizeof(float));
        hipMalloc(&m_density_previous_cuda, g_point_count * g_point_count * sizeof(float));
        hipMalloc(&m_u_cuda, g_point_count * g_point_count * sizeof(float));
        hipMalloc(&m_v_cuda, g_point_count * g_point_count * sizeof(float));
        hipMalloc(&m_u_previous_cuda, g_point_count * g_point_count * sizeof(float));
        hipMalloc(&m_v_previous_cuda, g_point_count * g_point_count * sizeof(float));
    }

    Fluid::~Fluid() {
        hipFree(m_density_cuda);
        hipFree(m_density_previous_cuda);
        hipFree(m_u_cuda);
        hipFree(m_v_cuda);
        hipFree(m_u_previous_cuda);
        hipFree(m_v_previous_cuda);
    }

    void Fluid::add_density(float x, float y, float multiplier) {
        if (x <= 0 || x >= 1 || y <= 0 || y >= 1)
            return;
        const auto grid_position = screen_to_array_indices(x, y);
        add_around_point_kernel<<<g_blocks, g_threads>>>(
            m_density_cuda, grid_position.i, grid_position.j, multiplier * g_particle_input, 0.045);
        hipDeviceSynchronize();
    }

    void Fluid::add_velocity(float x, float y, float dx, float dy) {
        if (x <= 0 || x >= 1 || y <= 0 || y >= 1)
            return;
        const auto grid_position = screen_to_array_indices(x, y);
        add_around_point_kernel<<<g_blocks, g_threads>>>(
            m_u_previous_cuda, grid_position.i, grid_position.j, g_force_input * dx, 0.045);
        add_around_point_kernel<<<g_blocks, g_threads>>>(
            m_v_previous_cuda, grid_position.i, grid_position.j, g_force_input * dy, 0.045);
        hipDeviceSynchronize();
    }

    float Fluid::sample_density_at(float x, float y) const {
        return m_density.sample_at(x, y);
    }

    float Fluid::sample_u_at(float x, float y) const {
        return m_u.sample_at(x, y);
    }

    float Fluid::sample_v_at(float x, float y) const {
        return m_v.sample_at(x, y);
    }

    void Fluid::clear_previous() {
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_previous_cuda);
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_previous_cuda);
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_previous_cuda);
        hipDeviceSynchronize();
    }

    void Fluid::clear_current() {
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_density_cuda);
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_v_cuda);
        set_to_zero_kernel<<<g_blocks, g_threads>>>(m_u_cuda);
        hipDeviceSynchronize();
    }

    void Fluid::density_step() {
        add_scaled(m_density_cuda, m_density_previous_cuda, g_dt);
        diffuse(m_density_previous_cuda, m_density_cuda, g_dt);
        advect(m_density_cuda, m_density_previous_cuda, m_u_cuda, m_v_cuda);

        decrease_density_kernel<<<g_blocks, g_threads>>>(m_density_cuda);
        hipDeviceSynchronize();
    }

    void Fluid::velocity_step() {
        add_scaled(m_u_cuda, m_u_previous_cuda, g_dt);
        add_scaled(m_v_cuda, m_v_previous_cuda, g_dt);
        diffuse(m_u_previous_cuda, m_u_cuda, g_viscosity_coefficient);
        diffuse(m_v_previous_cuda, m_v_cuda, g_viscosity_coefficient);
        project(m_u_previous_cuda, m_v_previous_cuda, m_u_cuda, m_v_cuda);
        advect(m_u_cuda, m_u_previous_cuda, m_u_previous_cuda, m_v_previous_cuda);
        advect(m_v_cuda, m_v_previous_cuda, m_u_previous_cuda, m_v_previous_cuda);
        project(m_u_cuda, m_v_cuda, m_u_previous_cuda, m_v_previous_cuda);
    }

    void Fluid::step() {
        velocity_step();
        density_step();
        get_from_gpu();
        clear_previous();
    }

    void Fluid::get_from_gpu() {
        hipMemcpy(m_density.data(), m_density_cuda, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(m_density_previous.data(),
                   m_density_previous_cuda,
                   g_point_count * g_point_count * sizeof(float),
                   hipMemcpyDeviceToHost);
        hipMemcpy(m_u.data(), m_u_cuda, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(m_v.data(), m_v_cuda, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(m_u_previous.data(), m_u_previous_cuda, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(m_v_previous.data(), m_v_previous_cuda, g_point_count * g_point_count * sizeof(float), hipMemcpyDeviceToHost);
    }

    void Fluid::set_polygon(size_t n, double intensity, math::Point direction_offset) {
        assert(n >= 3);
        assert(intensity >= 0 && intensity <= 1);
        std::vector<math::Point> points;
        for (size_t i = 0; i != n + 1; ++i) {
            float theta = i * 2.0f * M_PI / static_cast<float>(n);
            points.push_back(math::Point{0.5, 0.5} + 0.4 * intensity * math::Point{std::sin(theta), std::cos(theta)});
        }

        for (size_t i = 0; i != n; ++i) {
            const auto& point_1     = points[i];
            const auto& point_2     = points[i + 1];
            const auto  lerp_points = 41;
            for (size_t j = 0; j != lerp_points + 1; ++j) {
                const auto point = (j * point_1 + (lerp_points - j) * point_2) * (1.0 / static_cast<float>(lerp_points));
                add_density(point, intensity);
                add_velocity(point, 1.0 * intensity * intensity * intensity * (math::random_point_centered() + direction_offset));
            }
        }
    }

    const Matrix& Fluid::u() const {
        return m_u;
    }

    const Matrix& Fluid::v() const {
        return m_v;
    }

    const Matrix& Fluid::density() const {
        return m_density;
    }

    void Fluid::add_density(const math::Point& point, float multiplier) {
        add_density(point.x, point.y, multiplier);
    }

    void Fluid::add_velocity(const math::Point& point, const math::Direction& direction) {
        add_velocity(point.x, point.y, direction.x, direction.y);
    }

    /******************** Static function implementations ************************/

    static float clamp_to_zero_one(float val) {
        return val < 0.0f ? 0.0f : val > 1.0f ? 1.0f : val;
    }

    static Index screen_to_array_indices(float x, float y) {
        const auto i = static_cast<uint16_t>(clamp_to_zero_one(x) * (g_point_count - 1));
        const auto j = static_cast<uint16_t>(clamp_to_zero_one(y) * (g_point_count - 1));
        return {i, j};
    }

    static void advect(float* current, const float* previous, const float* u, const float* v) {
        advect_kernel<<<g_blocks, g_threads>>>(current, previous, u, v, g_dt * static_cast<float>(g_cell_count - 1));
        hipDeviceSynchronize();

        set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
        hipDeviceSynchronize();
    }

    static void diffuse(float* current, const float* previous, float weight) {
        float ratio = weight * g_diffusion_coefficient * (g_cell_count - 1.0f) * (g_cell_count - 1.0f) / g_point_count;
        for (size_t k = 0; k != 20; ++k) {
            diffuse_kernel<<<g_blocks, g_threads>>>(current, previous, ratio);
            hipDeviceSynchronize();
        }

        set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(current);
        hipDeviceSynchronize();
    }

    static void add_scaled(float* target, const float* addend, const float multiplier) {
        add_scaled_kernel<<<g_blocks, g_threads>>>(target, addend, multiplier);
        hipDeviceSynchronize();
    }

    static void project(float* u_gpu, float* v_gpu, float* u_previous_gpu, float* v_previous_gpu) {
        set_to_zero_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
        hipDeviceSynchronize();

        project_step_1_kernel<<<g_blocks, g_threads>>>(v_previous_gpu, u_gpu, v_gpu);
        hipDeviceSynchronize();

        for (size_t k = 0; k != 20; ++k) {
            project_step_2_kernel<<<g_blocks, g_threads>>>(u_previous_gpu);
            hipDeviceSynchronize();

            add_scaled(u_previous_gpu, v_previous_gpu, 0.25);
        }

        project_step_3_kernel<<<g_blocks, g_threads>>>(u_gpu, v_gpu, u_previous_gpu);
        hipDeviceSynchronize();

        set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(u_gpu);
        set_bounds_to_zero_kernel<<<g_blocks, g_threads>>>(v_gpu);
        hipDeviceSynchronize();
    }

    /******************** CUDA kernel function implementations ************************/

    __device__ int diff(uint16_t a, uint16_t b) {
        return a >= b ? static_cast<int>(a - b) : -static_cast<int>(b - a);
    }

    __device__ float distance_kernel(uint16_t i_1, uint16_t j_1, uint16_t i_2, uint16_t j_2) {
        const int d_i = diff(i_1, i_2);
        const int d_j = diff(j_1, j_2);

        return std::sqrt(static_cast<float>(d_i * d_i + d_j * d_j)) / static_cast<float>(g_point_count);
    }

    __global__ void
    add_around_point_kernel(float* density, uint16_t i_target, uint16_t j_target, float multiplier, float radius) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || i == g_point_count || j == g_point_count)
            return;

        const auto distance = distance_kernel(i, j, i_target, j_target);
        if (distance > radius)
            return;

        density[index] += (radius - distance) * multiplier / radius;
    }

    __global__ void set_to_zero_kernel(float* data) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        data[index] = 0;
    }

    __global__ void decrease_density_kernel(float* density) {
        unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < g_n)
            density[index] *= 0.97;
    }

    __device__ uint32_t to_linear_index(uint16_t i, uint16_t j) {
        return i * g_point_count + j;
    }

    __global__ void advect_kernel(float* current, const float* previous, const float* u, const float* v, float ratio) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;

        const float x_temp    = static_cast<float>(i) - ratio * u[index];
        const float y_temp    = static_cast<float>(j) - ratio * v[index];
        const float upper     = static_cast<float>(g_cell_count) - 0.5f;
        const float x_new     = x_temp < 0.5f ? 0.5f : x_temp > upper ? upper : x_temp;
        const float y_new     = y_temp < 0.5f ? 0.5f : y_temp > upper ? upper : y_temp;
        const int   x_new_int = static_cast<int>(x_new);
        const int   y_new_int = static_cast<int>(y_new);
        const float dt_x      = x_new - static_cast<float>(x_new_int);
        const float dt_y      = y_new - static_cast<float>(y_new_int);
        current[index]        = (1 - dt_x) * ((1 - dt_y) * previous[to_linear_index(x_new_int, y_new_int)] +
                                       dt_y * previous[to_linear_index(x_new_int, y_new_int + 1)]) +
                         dt_x * ((1 - dt_y) * previous[to_linear_index(x_new_int + 1, y_new_int)] +
                                 dt_y * previous[to_linear_index(x_new_int + 1, y_new_int + 1)]);
    }

    __global__ void set_bounds_to_zero_kernel(float* data) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;
        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || i == g_point_count - 1 || j == g_point_count - 1)
            data[index] = 0;
    }

    __device__ float sum_neighbors_kernel(const float* data, uint16_t i, uint16_t j) {
        assert(i > 0 && j > 0);
        return data[to_linear_index(i - 1, j)] + data[to_linear_index(i + 1, j)] + data[to_linear_index(i, j - 1)] +
               data[to_linear_index(i, j + 1)];
    }

    __global__ void diffuse_kernel(float* current, const float* previous, float ratio) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
            return;

        current[index] = (previous[index] + ratio * sum_neighbors_kernel(current, i, j)) / (1 + 4 * ratio);
    }

    __global__ void add_scaled_kernel(float* target, const float* addend, const float multiplier) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        target[index] += multiplier * addend[index];
    }

    __device__ float horizontal_difference(const float* data, uint16_t i, uint16_t j) {
        return data[to_linear_index(i, j + 1)] - data[to_linear_index(i, j - 1)];
    }

    __device__ float vertical_difference(const float* source, uint16_t i, uint16_t j) {
        return source[to_linear_index(i + 1, j)] - source[to_linear_index(i - 1, j)];
    }

    __global__ void project_step_3_kernel(float* u_current, float* v_current, const float* u_previous) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
            return;

        u_current[to_linear_index(i, j)] -= 0.5f * vertical_difference(u_previous, i, j) / g_cell_length;
        v_current[to_linear_index(i, j)] -= 0.5f * horizontal_difference(u_previous, i, j) / g_cell_length;
    }

    __global__ void project_step_2_kernel(float* u_previous) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
            return;

        u_previous[to_linear_index(i, j)] = sum_neighbors_kernel(u_previous, i, j) / 4.0f;
    }

    __global__ void project_step_1_kernel(float* v_previous, const float* u_current, const float* v_current) {
        const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= g_n)
            return;

        const unsigned i = index / g_point_count;
        const unsigned j = index % g_point_count;
        if (i == 0 || j == 0 || j >= g_point_count || i >= g_point_count)
            return;

        v_previous[index] =
            -0.5f * g_cell_length * (vertical_difference(u_current, i, j) + horizontal_difference(v_current, i, j));
    }

} // namespace app::fluid
